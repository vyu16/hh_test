/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2020 Victor Yu
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "hip/hip_runtime.h"

extern "C" {

int cuda_set_device(int i_gpu)
{
    hipError_t err = hipSetDevice(i_gpu);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaSetDevice: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_get_device_count(int *n_gpu)
{
    hipError_t err = hipGetDeviceCount(n_gpu);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaGetDeviceCount: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_device_synchronize()
{
    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess)
    {
        printf("\n Error in cudaDeviceSynchronize: %s \n", hipGetErrorString(err));
    }

    return 0;
}

int cuda_malloc(intptr_t *a, size_t size)
{
    hipError_t err = hipMalloc((void **) a, size);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaMalloc: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_free(intptr_t *a)
{
    hipError_t err = hipFree(a);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaFree: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_memcpy(intptr_t *dest, intptr_t *src, size_t count, int dir)
{
    hipMemcpyKind dir2;

    switch (dir)
    {
        case 0:
            dir2 = hipMemcpyHostToDevice;
            break;
        case 1:
            dir2 = hipMemcpyDeviceToHost;
            break;
        case 2:
            dir2 = hipMemcpyDeviceToDevice;
            break;
    }

    hipError_t err = hipMemcpy(dest, src, count, dir2);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaMemcpy: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

}
